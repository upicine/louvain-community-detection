#include "utils.cuh"
#include "modularity.cuh"
#include "aggregation.cuh"

int main(int argc, char **argv) {
    char *filename;
    unsigned int *v_set, *comm_set;
    float threshold, alg_time, copy_time;
    bool verbose = false;

    parseArgs(argc, argv, threshold, verbose, filename);

    Graph host_graph, dev_graph;
    CommAggr aggr_data;

    parseGraph(host_graph, filename);


    hipEvent_t copy_start, copy_stop, alg_start, alg_stop;
    HANDLE_ERROR(hipEventCreate(&copy_start));
    HANDLE_ERROR(hipEventCreate(&copy_stop));
    HANDLE_ERROR(hipEventRecord(copy_start, 0));

    initDevGraph(dev_graph, host_graph.vert_sz, host_graph.edges_sz);
    copyToDevice(dev_graph, host_graph);
    initCommAggr(aggr_data, dev_graph);
    setInitComm(dev_graph);

    HANDLE_ERROR(hipMalloc((void **) &v_set, dev_graph.vert_sz * sizeof(unsigned int)));
    HANDLE_ERROR(hipMalloc((void **) &comm_set, dev_graph.vert_sz * sizeof(unsigned int)));

    HANDLE_ERROR(hipEventRecord(copy_stop, 0));
    HANDLE_ERROR(hipEventSynchronize(copy_stop));
    HANDLE_ERROR(hipEventElapsedTime(&copy_time, copy_start, copy_stop));
    HANDLE_ERROR(hipEventDestroy(copy_start));
    HANDLE_ERROR(hipEventDestroy(copy_stop));


    HANDLE_ERROR(hipEventCreate(&alg_start));
    HANDLE_ERROR(hipEventCreate(&alg_stop));
    HANDLE_ERROR(hipEventRecord(alg_start, 0));

    for (;;) {
        if (!modularityOptimization(dev_graph, v_set, threshold)) {
            break;
        }
        aggregation(dev_graph, aggr_data, comm_set);
    }

    HANDLE_ERROR(hipEventRecord(alg_stop, 0));
    HANDLE_ERROR(hipEventSynchronize(alg_stop));
    HANDLE_ERROR(hipEventElapsedTime(&alg_time, alg_start, alg_stop));
    HANDLE_ERROR(hipEventDestroy(alg_start));
    HANDLE_ERROR(hipEventDestroy(alg_stop));

    std::cout << alg_time << " " << alg_time + copy_time << std::endl;

    if (verbose) {
        printCommunities(dev_graph);
    }

    HANDLE_ERROR(hipFree(v_set));
    HANDLE_ERROR(hipFree(comm_set));

    deleteHostGraph(host_graph);
    deleteDevGraph(dev_graph);
    deleteAggrData(aggr_data);

    return 0;
}
